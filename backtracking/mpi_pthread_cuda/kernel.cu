#include <hip/hip_runtime.h>


#include <stdio.h>

__global__ 
void n_queen_in_gpu( const short n_queens, const short first, const short second, int * global_ans, int * find_ans_local )
{
    short third = blockIdx.x;
    short forth = threadIdx.x;
    
    short * my;
    short * md1;
    short * md2;
    int * local_ans;
    
    short d1, d2;
    short o_d1, o_d2, o_p;
    short current_column;
    
    // allocate memory
    my = ( short * ) malloc ( sizeof ( short ) * n_queens ); 
    md1 = ( short * ) malloc ( sizeof ( short ) * n_queens * 2 );
    md2 = ( short * ) malloc ( sizeof ( short ) * n_queens * 2 );
    local_ans = ( int * ) malloc ( sizeof ( int ) * n_queens );
    
    // set the default value
    current_column = 4;
    memset(my, 0, sizeof(short) * n_queens);
    memset(md1, 0, sizeof(short) * n_queens * 2);
    memset(md2, 0, sizeof(short) * n_queens * 2);
    memset(local_ans, -1, sizeof(int) * n_queens);
    
    // filled the status finish in cpu
    my[first] = 1;
    my[second] = 1;
    
    md1[first] = 1;
    md1[( 1 + second ) % ( n_queens * 2 - 1 )] = 1;

    md2[ ( 0 - first + ( n_queens - 1) ) % ( n_queens * 2 - 1 ) ] = 1;
    md2[ ( 1 - second + ( n_queens - 1) ) % ( n_queens * 2 - 1 ) ] = 1;
    
    local_ans[0] = first;
    local_ans[1] = second;
        
    // test for chance of third place
    d1 = ( 2 + third ) % ( n_queens * 2 - 1 );
    d2 = ( 2 - third + ( n_queens - 1) ) % ( n_queens * 2 - 1 );
    
    if ( my[third] || md1[d1] || md2[d2] )
    {
        free( my );
        free( md1 );
        free( md2 );
        return;
    }
    
    my[third] = md1[d1] = md2[d2] = 1;
    local_ans[2] = third;
    
    // test for chance of forth place
    d1 = ( 3 + forth ) % ( n_queens * 2 - 1 );
    d2 = ( 3 - forth + ( n_queens - 1) ) % ( n_queens * 2 - 1 );
    
    if ( my[forth] || md1[d1] || md2[d2] )
    {
        free( my );
        free( md1 );
        free( md2 );
        return;
    }
    
    my[forth] = md1[d1] = md2[d2] = 1;
    local_ans[3] = forth;
    
    // check if somebody yet found the ans.
    while ( !( *find_ans_local ) )
    {
        // FIND ANS!!!!!
        if ( current_column == n_queens ) 
        {
            // notify to all gpu thread that I found the ANS!!!!!
            *find_ans_local = 1;
            
            memcpy( global_ans, local_ans, sizeof(int) * n_queens );
        }
        
        // first come into this column
        // there is no valid o_d1 and o_d2
        if ( local_ans[current_column] == -1 )
        {
            o_d1 = -1;
            o_d2 = -1;
            o_p = -1;
        }
        else
        {
            o_d1 = ( current_column + local_ans[current_column] ) % ( n_queens * 2 - 1 );
            o_d2 = ( current_column - local_ans[current_column] + ( n_queens - 1) ) % ( n_queens * 2 - 1 );
            o_p = local_ans[current_column];
        }
        
        // try the next possible ans
        do
        {
            local_ans[current_column] ++;
        
            if ( local_ans[current_column] < n_queens )
            {
                d1 = ( current_column + local_ans[current_column] ) % ( n_queens * 2 - 1 );
                d2 = ( current_column - local_ans[current_column] + ( n_queens - 1) ) % ( n_queens * 2 - 1 );    
            }
            else
            {
                break;
            }
            
        } while ( ( my[ local_ans[ current_column ] ] || md1[ d1 ] || md2[ d2 ] ) );
        
        // remove the mark of old position
        if ( o_d1 != -1 && o_d2 != -1 && o_p != -1 )
            my[ o_p ] = md1[ o_d1 ] = md2[ o_d2 ] = 0;
        
        // find the possible ans
        if ( local_ans[current_column] < n_queens )
        {
            // mark the board according to current position
            my[ local_ans[ current_column ] ] = md1[ d1 ] = md2[ d2 ] = 1;
            
            // go to next column
            current_column ++ ;
        }
        else
        {   
            local_ans[ current_column ] = -1;
            
            if ( current_column == 4 )
                break;
            else
                current_column --;
        }
    }
    
    free( my ); 
    free( md1 );
    free( md2 );
}

extern "C" int find_ans_in_gpu ( int * board, int n_queens )
{
    int * global_gpu_ans;
    int * find_ans_in_local_thread;
    
    hipMallocManaged( ( void** )&global_gpu_ans, sizeof( int ) * n_queens );
    hipMallocManaged( ( void** )&find_ans_in_local_thread, sizeof( int ) );
    
    ( *find_ans_in_local_thread ) = 0;
    
    n_queen_in_gpu <<< n_queens, n_queens >>> ( n_queens, board[0], board[1], global_gpu_ans, find_ans_in_local_thread );
    hipDeviceSynchronize();
    
    if ( *find_ans_in_local_thread )
    {        
        memcpy( board, global_gpu_ans, sizeof( int ) * n_queens );
        
        return 1;
    }
    else
    {
        return 0;
    }
}